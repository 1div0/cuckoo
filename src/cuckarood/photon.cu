#include "hip/hip_runtime.h"
// Cuckarood Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018-2019 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include "cuckarood.hpp"
#include "graph.hpp"
#include "../crypto/siphash.cuh"
#include "../crypto/blake2.h"
#include "kernel.cu"

typedef uint8_t u8;
typedef uint16_t u16;

#ifndef IDXSHIFT
// number of bits of compression of surviving edge endpoints
// reduces space used in cycle finding, but too high a value
// results in NODE OVERFLOW warnings and fake cycles
#define IDXSHIFT 12
#endif

const u32 MAXEDGES = NEDGES2 >> IDXSHIFT;

#ifndef NEPS_A
#define NEPS_A 134 // to match Photon's kernel.cu
#endif
#ifndef NEPS_B
#define NEPS_B 85 // to match Photon's kernel.cu
#endif
#define NEPS 128

const u32 EDGES_A = NZ * NEPS_A / NEPS;
const u32 EDGES_B = NZ * NEPS_B / NEPS;

const u32 ROW_EDGES_A = EDGES_A * NY;
const u32 ROW_EDGES_B = EDGES_B * NY;

// Number of Parts of BufferB, all but one of which will overlap BufferA
#ifndef NA
#define NA 4
#endif
#define NA2 (NA * NA)

__constant__ uint2 recoveredges[PROOFSIZE];
__constant__ uint2 e0 = {0,0};

__device__ u64 dipblock(const siphash_keys &keys, const word_t edge, u64 *buf) {
  diphash_state<25> shs(keys);
  word_t edge0 = edge & ~EDGE_BLOCK_MASK;
  u32 i;
  for (i=0; i < EDGE_BLOCK_MASK; i++) {
    shs.hash24(edge0 + i);
    buf[i] = shs.xor_lanes();
  }
  shs.hash24(edge0 + i);
  buf[i] = 0;
  return shs.xor_lanes();
}

__device__ u32 endpoint(uint2 nodes, int uorv) {
  return uorv ? nodes.y : nodes.x;
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template parameter
#define FLUSHA 16
#endif

template<int maxOut>
__global__ void SeedA(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, u32 * __restrict__ indexes) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int gid = group * dim + lid;
  const int nthreads = gridDim.x * dim;
  const int FLUSHA2 = 2*FLUSHA;

  __shared__ uint2 tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(uint2);
  __shared__ int counters[NX];
  u64 buf[EDGE_BLOCK_SIZE];

  for (int row = lid; row < NX; row += dim)
    counters[row] = 0;
  __syncthreads();

  const int col = group % NX;
  const int loops = NEDGES2 / nthreads; // assuming THREADS_HAVE_EDGES checked
  for (int blk = 0; blk < loops; blk += EDGE_BLOCK_SIZE) {
    u32 nonce0 = gid * loops + blk;
    const u64 last = dipblock(sipkeys, nonce0, buf);
    for (u32 e = 0; e < EDGE_BLOCK_SIZE; e++) {
      u64 edge = buf[e] ^ last;
      u32 dir = e & 1;
      u32 node0 = (edge        &  NODE1MASK) << 1 | dir;
      u32 node1 = (edge >> 31) & (NODE1MASK << 1) | dir;
      int row = node0 >> YZBITS;
      int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1)); // assuming ROWS_LIMIT_LOSSES checked
      tmp[row][counter] = make_uint2(node0, node1);
      __syncthreads();
      if (counter == FLUSHA-1) {
        int localIdx = min(FLUSHA2, counters[row]);
        int newCount = localIdx % FLUSHA;
        int nflush = localIdx - newCount;
        u32 grp = row * NX + col;
        int cnt = min((int)atomicAdd(indexes + grp, nflush), (int)(maxOut - nflush));
        for (int i = 0; i < nflush; i += TMPPERLL4)
          buffer[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
        for (int t = 0; t < newCount; t++) {
          tmp[row][t] = tmp[row][t + nflush];
        }
        counters[row] = newCount;
      }
      __syncthreads();
    }
  }
  uint2 zero = make_uint2(0, 0);
  for (int row = lid; row < NX; row += dim) {
    int localIdx = min(FLUSHA2, counters[row]);
    u32 grp = row * NX + col;
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[row][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(indexes + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      buffer[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
    }
  }
}

template <typename Edge> __device__ bool null(Edge e);

__device__ bool null(u32 nonce) {
  return nonce == 0;
}

__device__ bool null(uint2 nodes) {
  return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template<int maxOut>
__global__ void SeedB(const uint2 * __restrict__ source, ulonglong4 * __restrict__ destination, const u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int FLUSHB2 = 2 * FLUSHB;

  __shared__ uint2 tmp[NX][FLUSHB2];
  const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(uint2);
  __shared__ int counters[NX];

  for (int col = lid; col < NX; col += dim)
    counters[col] = 0;
  __syncthreads();
  const int row = group / NX;
  const int bucketEdges = min((int)srcIdx[group], (int)maxOut);
  const int loops = (bucketEdges + dim-1) / dim;
  for (int loop = 0; loop < loops; loop++) {
    int col;
    int counter = 0;
    const int edgeIndex = loop * dim + lid;
    if (edgeIndex < bucketEdges) {
      const int index = group * maxOut + edgeIndex;
      uint2 edge = __ldg(&source[index]);
      if (!null(edge)) {
        u32 node1 = edge.x;
        col = (node1 >> ZBITS) & XMASK;
        counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1)); // assuming COLS_LIMIT_LOSSES checked
        tmp[col][counter] = edge;
        }
    }
    __syncthreads();
    if (counter == FLUSHB-1) {
      int localIdx = min(FLUSHB2, counters[col]);
      int newCount = localIdx % FLUSHB;
      int nflush = localIdx - newCount;
      u32 grp = row * NX + col;
#ifdef SYNCBUG
      if (grp==0x2d6) printf("group %x size %d lid %d nflush %d\n", group, bucketEdges, lid, nflush);
#endif
      int cnt = min((int)atomicAdd(dstIdx + grp, nflush), (int)(maxOut - nflush));
      for (int i = 0; i < nflush; i += TMPPERLL4)
        destination[((u64)grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
      for (int t = 0; t < newCount; t++) {
        tmp[col][t] = tmp[col][t + nflush];
      }
      counters[col] = newCount;
    }
    __syncthreads(); 
  }
  uint2 zero = make_uint2(0, 0);
  for (int col = lid; col < NX; col += dim) {
    int localIdx = min(FLUSHB2, counters[col]);
    u32 grp = row * NX + col;
#ifdef SYNCBUG
    if (group==0x2f2 && grp==0x2d6) printf("group %x size %d lid %d localIdx %d\n", group, bucketEdges, lid, localIdx);
#endif
    for (int j = localIdx; j % TMPPERLL4; j++)
      tmp[col][j] = zero;
    for (int i = 0; i < localIdx; i += TMPPERLL4) {
      int cnt = min((int)atomicAdd(dstIdx + grp, TMPPERLL4), (int)(maxOut - TMPPERLL4));
      destination[((u64)grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
    }
  }
}

__device__ __forceinline__  void bitmapset(u32 *ebitmap, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;
  atomicOr(ebitmap + word, mask);
}

__device__ __forceinline__  bool bitmaptest(u32 *ebitmap, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  return (ebitmap[word] >> bit) & 1;
}

template<int NP, int maxIn, int maxOut>
__global__ void Round(const int round, const uint2 * __restrict__ src, uint2 * __restrict__ dst, const u32 * __restrict__ srcIdx, u32 * __restrict__ dstIdx) {
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  const int lid = threadIdx.x;
  const int BITMAPWORDS = NZ / 32;

  __shared__ u32 ebitmap[BITMAPWORDS];

  for (int i = lid; i < BITMAPWORDS; i += dim)
    ebitmap[i] = 0;
  __syncthreads();

  for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
    const int edgesInBucket = min(srcIdx[group], maxIn);
    // if (!group && !lid) printf("round %d size  %d\n", round, edgesInBucket);
    const int loops = (edgesInBucket + dim-1) / dim;

    for (int loop = 0; loop < loops; loop++) {
      const int lindex = loop * dim + lid;
      if (lindex < edgesInBucket) {
        const int index = maxIn * group + lindex;
        uint2 edge = __ldg(&src[index]);
        if (null(edge)) continue;
        u32 node = endpoint(edge, round&1);
        bitmapset(ebitmap, node & ZMASK);
      }
    }
  }

  __syncthreads();

  src -= NP * NX2 * maxIn; srcIdx -= NP * NX2;
  for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
    const int edgesInBucket = min(srcIdx[group], maxIn);
    const int loops = (edgesInBucket + dim-1) / dim;
    for (int loop = 0; loop < loops; loop++) {
      const int lindex = loop * dim + lid;
      if (lindex < edgesInBucket) {
        const int index = maxIn * group + lindex;
        uint2 edge = __ldg(&src[index]);
        if (null(edge)) continue;
        u32 node0 = endpoint(edge, round&1);
        if (bitmaptest(ebitmap, (node0 & ZMASK) ^ 1)) {
          u32 node1 = endpoint(edge, (round&1)^1);
          const int bucket = node1 >> ZBITS;
          const int bktIdx = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
          dst[bucket * maxOut + bktIdx] = (round&1) ? make_uint2(node1, node0) : make_uint2(node0, node1);
        }
      }
    }
  }
}

template<int maxIn>
__global__ void Tail(const uint2 *source, uint2 *destination, const u32 *srcIdx, u32 *dstIdx) {
  const int lid = threadIdx.x;
  const int group = blockIdx.x;
  const int dim = blockDim.x;
  int myEdges = srcIdx[group];
  __shared__ int destIdx;

  if (lid == 0)
    destIdx = atomicAdd(dstIdx, myEdges);
  __syncthreads();
  for (int i = lid; i < myEdges; i += dim)
    destination[destIdx + lid] = source[group * maxIn + lid];
}

#define checkCudaErrors_V(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return;})
#define checkCudaErrors_N(ans) ({if (gpuAssert((ans), __FILE__, __LINE__) != hipSuccess) return NULL;})
#define checkCudaErrors(ans) ({int retval = gpuAssert((ans), __FILE__, __LINE__); if (retval != hipSuccess) return retval;})

inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
  if (code != hipSuccess) {
    snprintf(LAST_ERROR_REASON, MAX_NAME_LEN, "Device %d GPUassert: %s %s %d", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    if (abort) return code;
  }
  return code;
}

__global__ void Recovery(const siphash_keys &sipkeys, ulonglong4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES2 / nthreads;
  __shared__ u32 nonces[PROOFSIZE];
  u64 buf[EDGE_BLOCK_SIZE];

  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int blk = 0; blk < loops; blk += EDGE_BLOCK_SIZE) {
    u32 nonce0 = gid * loops + blk;
    const u64 last = dipblock(sipkeys, nonce0, buf);
    for (int i = 0; i < EDGE_BLOCK_SIZE; i++) {
      u64 edge = buf[i] ^ last;
      u32 dir = i & 1;
      u32 u = (edge & NODE1MASK) << 1 | dir;
      u32 v = ((edge >> 32) & NODE1MASK) << 1 | dir;
      for (int p = 0; p < PROOFSIZE; p++) { //YO
        if (recoveredges[p].x == u && recoveredges[p].y == v) {
          nonces[p] = nonce0 + i;
        }
      }
    }
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
    if (nonces[lid] > 0)
      indexes[lid] = nonces[lid];
  }
}

struct blockstpb {
  u16 blocks;
  u16 tpb;
};

#ifndef SEED_TPB
#define SEED_TPB 256
#endif
#ifndef TRIM0_TPB
#define TRIM0_TPB 1024
#endif
#ifndef TRIM1_TPB
#define TRIM1_TPB 512
#endif
#ifndef TRIM_TPB
#define TRIM_TPB 512
#endif

struct trimparams {
  u16 ntrims;
  blockstpb seed;
  blockstpb trim0;
  blockstpb trim1;
  blockstpb trim;
  blockstpb tail;
  blockstpb recover;

  trimparams() {
    ntrims         =       458;
    seed.blocks    =        64;
    seed.tpb       =  SEED_TPB;
    trim0.blocks   =    NX2/NA;
    trim0.tpb      = TRIM0_TPB;
    trim1.blocks   =    NX2/NA;
    trim1.tpb      = TRIM1_TPB;
    trim.blocks    =       NX2;
    trim.tpb       =  TRIM_TPB;
    tail.blocks    =       NX2;
    tail.tpb       =       256;
    recover.blocks =      2048;
    recover.tpb    =       256;
  }
};

typedef u32 proof[PROOFSIZE];

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  edgetrimmer *dt;
  size_t sizeA, sizeB;
  const size_t indexesSize = NX2 * sizeof(u32);
  const size_t indexesSizeNA = NA * indexesSize;
  u8 *bufferA;
  u8 *bufferB;
  u8 *bufferA1;
  u32 *indexesA;
  u32 *indexesB;
  u32 nedges;
  u32 *uvnodes;
  siphash_keys sipkeys;
  bool abort;
  bool initsuccess = false;

  edgetrimmer(const trimparams _tp) : tp(_tp) {
    checkCudaErrors_V(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors_V(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
    checkCudaErrors_V(hipMalloc((void**)&indexesA, indexesSizeNA));
    checkCudaErrors_V(hipMalloc((void**)&indexesB, indexesSizeNA));
    sizeA = ROW_EDGES_A * NX * sizeof(uint2);
    sizeB = ROW_EDGES_B * NX * sizeof(uint2);
    const size_t bufferSize = sizeA + sizeB / NA;
    checkCudaErrors_V(hipMalloc((void**)&bufferB, bufferSize));
    bufferA = bufferB + sizeB / NA;
    bufferA1 = bufferB + sizeB;
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
  }
  u64 globalbytes() const {
    return (sizeA+sizeB/NA) + (1+NA) * indexesSize + sizeof(siphash_keys) + PROOFSIZE * 2 * sizeof(u32) + sizeof(edgetrimmer);
  }
  ~edgetrimmer() {
    checkCudaErrors_V(hipFree(bufferB));
    checkCudaErrors_V(hipFree(indexesA));
    checkCudaErrors_V(hipFree(indexesB));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
  }
  u32 trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));
    hipMemcpyToSymbol(HIP_SYMBOL(dipkeys), &sipkeys, sizeof(sipkeys));

    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);
  

    hipMemset(indexesA, 0, indexesSizeNA);
    for (u32 i=0; i < NA; i++) {
      FluffySeed4K<SEED_TPB, EDGES_A/NA><<<tp.seed.blocks, tp.seed.tpb>>>((uint4*)(bufferA+i*(sizeA/NA2)), indexesA+i*NX2, i*(NEDGES2/NA));
      if (abort) return false;
    }
  
#ifdef VERBOSE
    print_log("%d x Seed4K<<<%d,%d>>>\n", NA, tp.seed.blocks, tp.seed.tpb); // 1024x512
    hipMemcpy(&nedges, indexesA, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("round %d edges %d\n", 0, nedges);
#endif

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop);
    hipEventRecord(start, NULL);
  
    hipMemset(indexesB, 0, indexesSizeNA);
    const u32 qB = sizeB/NA;
    const u32 qI = NX2 / NA;
    for (u32 i=0; i < NA; i++) {
      FluffyRound_A1<TRIM0_TPB, EDGES_A/NA, EDGES_B/NA><<<NX2/NA, TRIM0_TPB>>>((uint2*)bufferA, (uint4*)(bufferB+i*qB), indexesA, indexesB, i*qI);
      if (abort) return false;
    }

#ifdef VERBOSE
    print_log("Seeding completed in %.0f ms\n", durationA);
    print_log("Round_A1<<<%d,%d>>>\n", NX2/NA, TRIM0_TPB); // 1024x1024
    hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("round %d edges %d\n", 1, nedges);
#endif

    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
    checkCudaErrors(hipEventDestroy(start)); checkCudaErrors(hipEventDestroy(stop));
    // print_log("Round 0 completed in %.0f ms\n", durationB);
  
    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A3<TRIM1_TPB, EDGES_B/NA, EDGES_B/2><<<NX2, TRIM1_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA);
    if (abort) return false;

#ifdef VERBOSE
    print_log("Round_A3<<<%d,%d>>>\n", NX2/NA, TRIM1_TPB); // 4096x1024
    hipMemcpy(&nedges, indexesA, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("round %d edges %d\n", 1, nedges);
#endif

    hipMemset(indexesB, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/2, EDGES_B/2><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, 2, 0);
    if (abort) return false;

#ifdef VERBOSE
    print_log("Round_A2<><<<%d,%d>>>\n", NX2, TRIM_TPB); // 4096x512
    hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("round %d edges %d\n", 2, nedges);
#endif

    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/2, EDGES_B/2><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, 3, 0);
    if (abort) return false;

    // hipMemcpy(&nedges, indexesA, sizeof(u32), hipMemcpyDeviceToHost);
    // hipDeviceSynchronize();
    // print_log("round %d edges %d\n", 3, nedges);

    hipMemset(indexesB, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/2, EDGES_B/2><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, 4, 0);
    if (abort) return false;

    // hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    // hipDeviceSynchronize();
    // print_log("round %d edges %d\n", 4, nedges);

    hipMemset(indexesA, 0, indexesSize);
    FluffyRound_A2<TRIM_TPB, EDGES_B/2, EDGES_B/4><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, 5, 0);
    if (abort) return false;

    // hipMemcpy(&nedges, indexesA, sizeof(u32), hipMemcpyDeviceToHost);
    // hipDeviceSynchronize();
    // print_log("round %d edges %d\n", 5, nedges);

    hipDeviceSynchronize();
  
    for (int round = 6; round < tp.ntrims; round += 2) {
      hipMemset(indexesB, 0, indexesSize);
      FluffyRound_A2<TRIM_TPB, EDGES_B/4, EDGES_B/4><<<NX2, TRIM_TPB>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB, round, 0);
      if (abort) return false;

      hipMemset(indexesA, 0, indexesSize);
      FluffyRound_A2<TRIM_TPB, EDGES_B/4, EDGES_B/4><<<NX2, TRIM_TPB>>>((uint2*)bufferB, (uint2*)bufferA1, indexesB, indexesA, round+1, 0);
      if (abort) return false;
    }
    
    hipMemset(indexesB, 0, indexesSize);
    FluffyTail<EDGES_B/4><<<tp.tail.blocks, tp.tail.tpb>>>((uint2*)bufferA1, (uint2*)bufferB, indexesA, indexesB);

    hipMemcpy(&nedges, indexesB, sizeof(u32), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    print_log("%d rounds %d edges\n", tp.ntrims, nedges);
    return nedges;
  }
};

struct solver_ctx {
  edgetrimmer trimmer;
  bool mutatenonce;
  uint2 *edges;
  graph<word_t> cg;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices

  solver_ctx(const trimparams tp, bool mutate_nonce) : trimmer(tp), cg(MAXEDGES, MAXEDGES, MAX_SOLS, IDXSHIFT) {
    edges   = new uint2[MAXEDGES];
    mutatenonce = mutate_nonce;
  }

  void setheadernonce(char * const headernonce, const u32 len, const u32 nonce) {
    if (mutatenonce)
      ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &trimmer.sipkeys);
    sols.clear();
  }
  ~solver_ctx() {
    delete[] edges;
  }

  int findcycles(uint2 *edges, u32 nedges) {
    cg.reset();
    for (u32 i = 0; i < nedges; i++)
      cg.add_compress_edge(edges[i].x, edges[i].y);
    for (u32 s = 0 ;s < cg.nsols; s++) {
      // print_log("Solution");
      for (u32 j = 0; j < PROOFSIZE; j++) {
        soledges[j] = edges[cg.sols[s][j]];
        // print_log(" (%x, %x)", soledges[j].x>>1, soledges[j].y>>1);
      }
      // print_log("\n");
      sols.resize(sols.size() + PROOFSIZE);
      hipMemcpyToSymbol(HIP_SYMBOL(recovery), soledges, sizeof(soledges));
      hipMemset(trimmer.indexesA, 0, trimmer.indexesSize);
      FluffyRecovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>((u32 *)trimmer.indexesA);
      hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer.indexesA, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
      checkCudaErrors(hipDeviceSynchronize());
      qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), cg.nonce_cmp);
    }
    return 0;
  }

  int solve() {
    u64 time0, time1;
    u32 timems,timems2;

    trimmer.abort = false;
    time0 = timestamp();
    u32 nedges = trimmer.trim();
    if (!nedges)
      return 0;
    if (nedges > MAXEDGES) {
      print_log("OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
    }
    hipMemcpy(edges, trimmer.bufferB, sizeof(uint2[nedges]), hipMemcpyDeviceToHost);
    time1 = timestamp(); timems  = (time1 - time0) / 1000000;
    time0 = timestamp();
    findcycles(edges, nedges);
    time1 = timestamp(); timems2 = (time1 - time0) / 1000000;
    print_log("trim time %d ms findcycles edges %d time %d ms total %d ms\n", timems, nedges, timems2, timems+timems2);
    return sols.size() / PROOFSIZE;
  }

  void abort() {
    trimmer.abort = true;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

typedef solver_ctx SolverCtx;

CALL_CONVENTION int run_solver(SolverCtx* ctx,
                               char* header,
                               int header_length,
                               u32 nonce,
                               u32 range,
                               SolverSolutions *solutions,
                               SolverStats *stats
                               )
{
  u64 time0, time1;
  u32 timems;
  u32 sumnsols = 0;
  int device_id;
  if (stats != NULL) {
    hipGetDevice(&device_id);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, stats->device_id);
    stats->device_id = device_id;
    stats->edge_bits = EDGEBITS;
    strncpy(stats->device_name, props.name, MAX_NAME_LEN);
  }

  if (ctx == NULL || !ctx->trimmer.initsuccess){
    print_log("Error initialising trimmer. Aborting.\n");
    print_log("Reason: %s\n", LAST_ERROR_REASON);
    if (stats != NULL) {
       stats->has_errored = true;
       strncpy(stats->error_reason, LAST_ERROR_REASON, MAX_NAME_LEN);
    }
    return 0;
  }

  for (u32 r = 0; r < range; r++) {
    time0 = timestamp();
    ctx->setheadernonce(header, header_length, nonce + r);
    print_log("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r, ctx->trimmer.sipkeys.k0, ctx->trimmer.sipkeys.k1, ctx->trimmer.sipkeys.k2, ctx->trimmer.sipkeys.k3);
    u32 nsols = ctx->solve();
    time1 = timestamp();
    timems = (time1 - time0) / 1000000;
    print_log("Time: %d ms\n", timems);
    for (unsigned s = 0; s < nsols; s++) {
      print_log("Solution");
      u32* prf = &ctx->sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++)
        print_log(" %jx", (uintmax_t)prf[i]);
      print_log("\n");
      if (solutions != NULL){
        solutions->edge_bits = EDGEBITS;
        solutions->num_sols++;
        solutions->sols[sumnsols+s].nonce = nonce + r;
        for (u32 i = 0; i < PROOFSIZE; i++) 
          solutions->sols[sumnsols+s].proof[i] = (u64) prf[i];
      }
      int pow_rc = verify(prf, ctx->trimmer.sipkeys);
      if (pow_rc == POW_OK) {
        print_log("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          print_log("%02x", cyclehash[i]);
        print_log("\n");
      } else {
        print_log("FAILED due to %s\n", errstr[pow_rc]);
      }
    }
    sumnsols += nsols;
    if (stats != NULL) {
      stats->last_start_time = time0;
      stats->last_end_time = time1;
      stats->last_solution_time = time1 - time0;
    }
  }
  print_log("%d total solutions\n", sumnsols);
  return sumnsols > 0;
}

CALL_CONVENTION SolverCtx* create_solver_ctx(SolverParams* params) {
  trimparams tp;
  tp.ntrims = params->ntrims;
  tp.seed.blocks = params->genablocks;
  tp.seed.tpb = params->genatpb;
  tp.trim0.tpb = params->genbtpb;
  tp.trim.tpb = params->trimtpb;
  tp.tail.tpb = params->tailtpb;
  tp.recover.blocks = params->recoverblocks;
  print_log("create_solver_ctx %d = %d\n", tp.recover.tpb, params->recovertpb);
  tp.recover.tpb = params->recovertpb;

  hipDeviceProp_t prop;
  checkCudaErrors_N(hipGetDeviceProperties(&prop, params->device));

  assert(tp.seed.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim0.tpb <= prop.maxThreadsPerBlock);
  assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
  // assert(tp.tailblocks <= prop.threadDims[0]);
  assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
  assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

  assert(tp.seed.blocks * tp.seed.tpb * EDGE_BLOCK_SIZE <= NEDGES2); // check THREADS_HAVE_EDGES
  assert(tp.recover.blocks * tp.recover.tpb * EDGE_BLOCK_SIZE <= NEDGES2); // check THREADS_HAVE_EDGES

  hipSetDevice(params->device);
  if (!params->cpuload)
    checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  return new SolverCtx(tp, params->mutate_nonce);
}

CALL_CONVENTION void destroy_solver_ctx(SolverCtx* ctx) {
  delete ctx;
}

CALL_CONVENTION void stop_solver(SolverCtx* ctx) {
  ctx->abort();
}

CALL_CONVENTION void fill_default_params(SolverParams* params) {
  trimparams tp;
  params->device = 0;
  params->ntrims = tp.ntrims;
  params->genablocks = tp.seed.blocks;
  params->genatpb = tp.seed.tpb;
  params->genbtpb = tp.trim0.tpb;
  params->trimtpb = tp.trim.tpb;
  params->tailtpb = tp.tail.tpb;
  params->recoverblocks = tp.recover.blocks;
  params->recovertpb = tp.recover.tpb;
  params->cpuload = false;
}

int main(int argc, char **argv) {
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len;
  int c;

  // set defaults
  SolverParams params;
  fill_default_params(&params);

  memset(header, 0, sizeof(header));
  while ((c = getopt(argc, argv, "scd:h:m:n:r:U:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        print_log("SYNOPSIS\n  cuda%d [-s] [-c] [-d device] [-h hexheader] [-m trims] [-n nonce] [-r range] [-U seedblocks] [-y Tailthreads] [-Z recoverblocks] [-z recoverthreads]\n", EDGEBITS);
        print_log("DEFAULTS\n  cuda%d -d %d -h \"\" -m %d -n %d -r %d -U %d -y %d -Z %d -z %d\n", EDGEBITS, device, tp.ntrims, nonce, range, tp.seed.blocks, tp.tail.tpb, tp.recover.blocks, tp.recover.tpb);
        exit(0);
      case 'c':
        params.cpuload = false;
        break;
      case 'd':
        device = params.device = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i); // hh specifies storage of a single byte
        break;
      case 'm': // ntrims         =       458;
        params.ntrims = atoi(optarg) & -2; // odd number of trimming rounds is treated same as 1 less anyway
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U': // seed.blocks    =        64;
        params.genablocks = atoi(optarg);
        break;
      case 'y': // tail.tpb       =       256;
        params.tailtpb = atoi(optarg);
        break;
      case 'Z': // recover.blocks =      2048;
        params.recoverblocks = atoi(optarg);
        break;
      case 'z': // recover.tpb    =       256;
        params.recovertpb = atoi(optarg);
        break;
    }
  }

  int nDevices;
  checkCudaErrors(hipGetDeviceCount(&nDevices));
  assert(device < nDevices);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, device));
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 102040; dbytes>>=10,dunit++) ;
  print_log("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);
  // hipSetDevice(device);

  print_log("Looking for %d-cycle on cuckarood%d(\"%s\",%d", PROOFSIZE, EDGEBITS, header, nonce);
  if (range > 1)
    print_log("-%d", nonce+range-1);
  print_log(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, params.ntrims, NX);

  SolverCtx* ctx = create_solver_ctx(&params);

  u64 bytes = ctx->trimmer.globalbytes();
  int unit;
  for (unit=0; bytes >= 102400; bytes>>=10,unit++) ;
  print_log("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);

  run_solver(ctx, header, sizeof(header), nonce, range, NULL, NULL);

  return 0;
}
